#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <fstream>
#include<sstream>
#include<string.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <ctime>

using namespace std;
#define index(i, j, colNum)  ((i)*(colNum)) + (j)

#include "0.header.h"
#include "0.matrixCalculation.cpp"
#include "0.CUDAmatCalculation.cpp"

int main(int argc, char *argv[]) {
	// double time_taken;
 //   clock_t start, end;


	int rowNum = atoi(argv[1]);
	int colNum = atoi(argv[2]);

	float * X = (float *) calloc(rowNum*colNum, sizeof(float));
	float * transposeX = (float *) calloc(colNum*rowNum, sizeof(float));

	float * Y = (float *) calloc(rowNum, sizeof(float));
	readFile(X, Y, rowNum, colNum);

	// display(X, rowNum, colNum);
	// display(Y, rowNum, 1);

	for(int i = 0; i < rowNum; i++) {
		for(int j = 0; j < colNum; j++) {
			transposeX[index(j,i, rowNum)] = X[index(i,j, colNum)];	
		}
	}

	
	N = colNum;
	float * tranXmulxMat = (float *) calloc(colNum*colNum, sizeof(float));
	// matMultiplFloat(transposeX, colNum, rowNum, X, rowNum, colNum, tranXmulxMat);
	cudaMatMultiCall(transposeX, colNum, rowNum, X, rowNum, colNum, tranXmulxMat);

	// cout<<"X\'-------->"<<endl;
	// display(transposeX, colNum, rowNum);

	// cout<<"X ----------->"<<endl;
	// display(X, rowNum, colNum);
	
	// cout<<"X\'X----------->"<<endl;
	// display(tranXmulxMat, colNum, colNum);

	// /////////////////////////////////////////////////////////////////////
	float inv[N*N]; // To store inverse of A[][] 

	// start = clock();

	// display(tranXmulxMat, colNum, colNum);

	mainInverseCuda(tranXmulxMat, colNum, inv);
	// display(inv, colNum, colNum);

	// cout<<"The inverse is --------->"<<endl;
	// if (inverse(tranXmulxMat, inv)) {
	// 	// display(inv, colNum, colNum); 
	// }

	// end = clock();
	//cal (X'X)^-1*X'
	// ------------

	float resultM[colNum*rowNum];
	// matMultiplFloat(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);
	cudaMatMultiCall(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);
	
	// cout<<"(X'X)^-1*X' ->"<<endl;
	// display(resultM, colNum, rowNum);
	
	//cal (X'X)^-1*X'Y
	// cout<<"(X'X)^-1*X'Y-->"<<endl;
	float finalResult[colNum];
	// matMultiplFloat(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);
	cudaMatMultiCall(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);

	cout<<"final (X'X)^-1*X'Y ->"<<endl;
	display(finalResult, colNum, 1);

	// time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
	// cout<<" single Core Taken->"<<time_taken<<endl;
	
	return 0;
}
